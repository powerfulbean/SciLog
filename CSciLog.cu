#include "hip/hip_runtime.h"
/**********************************/

#include "CSciLog.h"


int printHello()
{
cout<<"hello"<<endl;
}

CSciLog::CSciLog(string s)
{
 //m_timeStart = clock();
 timeval temp;
 gettimeofday(pm_timeStart, NULL);
 //m_timeStart = temp;
 m_dir = s + ".log";
 m_file.open(m_dir.data(),std::fstream::in | std::fstream::out | std::fstream::app);
 m_file.close();
}

void CSciLog::open()
{
  m_file.open(m_dir.data(),std::fstream::in | std::fstream::out | std::fstream::app);
}	

void CSciLog::close()
{
  m_file.close();
}

void CSciLog::record(string s, bool nextLine)
{
  open();
  m_file<<s;
  if(nextLine == true)
  {
        m_file<<endl;
  }
  else
  {
  	m_file<<"\t";
  }
  close();
  return ;
}

void CSciLog::recordTime(string s, bool nextLine)
{
  open();
  clock_t time = clock();
  double dTime = (double)time/CLOCKS_PER_SEC;
  m_file<<dTime;
  if(s.size()!=0)
  {
    m_file<<"\t";
    m_file<<s;
  }
  if(nextLine == true)
  {
	m_file<<endl;
  }
  else
  {
  	m_file<<"\t";
  }
  close();
  return ;
}

void CSciLog::recordTimeReadable(string s, bool nextLine)
{
  open();
  clock_t time = clock();
  double dTime = (double)time/CLOCKS_PER_SEC;
  m_file<<"Time "<<dTime<<": ";
  m_file<<s;
  if(nextLine == true)
  {
	m_file<<endl;
  }
  else
  {
  	m_file<<"\t";
  }
  close();
  return ;
}

void CSciLog::recordTimeFast(string s, bool nextLine, bool readable)
{
  //open();
  //clock_t time = clock();
  //double dTime = (double)time/CLOCKS_PER_SEC;
  string temp;
  if(readable == false){
	temp = getTimePassedString();
  }
  else{
	  temp = "Time "+getTimePassedString()+": ";
	  temp+=s;
	  temp+=";";
  }
  if(nextLine == true)
  {
	temp += "\n";
  }
  else
  {
  	temp += "\t";
  }
  m_cache.push_back(temp);
  return ;
}

void CSciLog::recordTimeReadableFast(string s, bool nextLine)
{
  
  /*clock_t timeSS = clock();
  double dTime = (double)timeSS/(double)CLOCKS_PER_SEC;
  char timeS[50];
  //time_t tTest;
  //time(&tTest);
  //double time_taken = double(timeSS)/double(CLOCKS_PER_SEC);
  //cout<<dTime<<setprecision(5)<<" ";
  timeval curTime;
  gettimeofday(&curTime, NULL);
  timeval resTime;
  timersub(&curTime,pm_timeStart,&resTime);
  char currentTime[84] = "";
  int sec = resTime.tv_sec;
  int usec = resTime.tv_usec;
  sprintf(currentTime, "%d\.%d", sec,usec);
  //printf("current time: %s \n", currentTime);
  //sprintf(timeS,"%f", dTime); */
  
  string temp = "Time "+getTimePassedString()+": ";
  temp+=s;
  temp+=";";
  if(nextLine == true)
  {
	temp += "\n";
  }
  else
  {
  	temp += "\t";
  }
  m_cache.push_back(temp);
  return ;
}


void CSciLog::save()
{
	open();
	for (int i=0;i<m_cache.size();i++)
	{
		m_file<<m_cache[i];
	}
	m_cache.clear();
	close();	
}


string CSciLog::getTimePassedString()
{
  timeval curTime;
  gettimeofday(&curTime, NULL);
  timeval resTime;
  timersub(&curTime,pm_timeStart,&resTime);
  char currentTime[84] = "";
  int sec = resTime.tv_sec;
  int usec = resTime.tv_usec;
  double finalAns = double(sec) + (1.0*usec)/double(1000000);
  //if(usec==99999)
  //{
//	  printf("%.6f %.6f %.6f\n",double(sec),(1.0*usec)/double(1000000),finalAns);
 // }
  sprintf(currentTime, "%.6f", finalAns);
  return std::string(currentTime);
}
/*
int main()
{
  fstream a;
  //a.open(".//testLog1.txt", std::fstream::in | std::fstream::out | std::fstream::app);
  CSciLog oLog(".//testLog.log");
  oLog.recordTime();
}*/
 
